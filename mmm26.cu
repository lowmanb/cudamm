#include "hip/hip_runtime.h"
#include<stdio.h>
#include<sys/time.h>
#include<stdlib.h>
#include<iostream>
#include<string.h>

using namespace std;

//----------------------------------- Structures and Globals---------------------------------------------

typedef struct {
    int dimension1;
    int dimension2;
} ArrayMetadata2D;

// metadata variables describing dimensionalities of all data structures involved in the computation
ArrayMetadata2D A_MD, B_MD, C_MD;
// pointers for input and output arrays in the host memory
float *A, *B, *C, *C_CPU;
// pointers for input and output arrays in the device memory (NVIDIA DRAM)
float *A_GPU, *B_GPU, *C_GPU;

//----------------------------------- host function definitions -----------------------------------------

void allocateAndInitializeAB();
void computeCpuMMM();
__global__ void computeGPUMMM(float* A, float* B, float* C, int size);
void copyMatricesToGPU();
void copyResultFromGPU();
void compareHostAndGpuOutput();
void die(const char *error);
void check_error(hipError_t e);

//----------------------------------- CUDA function definitions -----------------------------------------
#define BLOCK_DIM 26
clock_t start, endt, mem_transfer, compute;

//-------------------------------------------------------------------------------------------------------
int main(int argc, char **argv) {

    A_MD.dimension1 = (argc > 1) ? atoi(argv[1]) : 100;
    A_MD.dimension2 = (argc > 2) ? atoi(argv[2]) : A_MD.dimension1;
    B_MD.dimension1 = (argc > 3) ? atoi(argv[3]) : A_MD.dimension2;
    B_MD.dimension2 = (argc > 4) ? atoi(argv[4]) : B_MD.dimension1;
    C_MD.dimension1 = A_MD.dimension1;
    C_MD.dimension2 = B_MD.dimension2;

    // pad dimensions to next multiple of BLOCK_DIM
    if (A_MD.dimension1 % BLOCK_DIM != 0)
        A_MD.dimension1 = ((int)A_MD.dimension1/(3*BLOCK_DIM) + 1) * 3*BLOCK_DIM;
    if (A_MD.dimension2 % BLOCK_DIM != 0)
        A_MD.dimension2 = ((int)A_MD.dimension2/(3*BLOCK_DIM) + 1) * 3*BLOCK_DIM; 
    if (B_MD.dimension1 % BLOCK_DIM != 0)
        B_MD.dimension1 = ((int)B_MD.dimension1/(3*BLOCK_DIM) + 1) * 3*BLOCK_DIM;
    if (B_MD.dimension2 % BLOCK_DIM != 0)
        B_MD.dimension2 = ((int)B_MD.dimension2/(3*BLOCK_DIM) + 1) * 3*BLOCK_DIM;
    if (C_MD.dimension1 % BLOCK_DIM != 0)
        C_MD.dimension1 = ((int)C_MD.dimension1/(3*BLOCK_DIM) + 1) * 3*BLOCK_DIM;
    if (C_MD.dimension2 % BLOCK_DIM != 0)
        C_MD.dimension2 = ((int)C_MD.dimension2/(3*BLOCK_DIM) + 1) * 3*BLOCK_DIM;

    printf("Matrix A is %d-by-%d\n", A_MD.dimension1, A_MD.dimension2);
    printf("Matrix B is %d-by-%d\n", B_MD.dimension1, B_MD.dimension2);
    printf("Matrix C is %d-by-%d\n", C_MD.dimension1, C_MD.dimension2);

    allocateAndInitializeAB();

    // matrix matrix multiplication in the CPU
    /*
    start = clock();
    computeCpuMMM();
    endt = clock();
    double elapsed = (endt - start) / (double) CLOCKS_PER_SEC;
    printf("Computation time in the CPU: %f seconds\n", elapsed);
    */

    // MMM on the GPU
    start = clock();
    copyMatricesToGPU();
    endt = clock();
    mem_transfer = endt - start;

    dim3 blocks(A_MD.dimension1/(3*BLOCK_DIM), A_MD.dimension1/(3*BLOCK_DIM));
    dim3 threads(BLOCK_DIM, BLOCK_DIM);

    start = clock();
    computeGPUMMM<<<blocks, threads>>>(A_GPU, B_GPU, C_GPU, A_MD.dimension1);
    hipDeviceSynchronize();
    endt = clock();
    compute = endt - start;

    check_error(hipGetLastError());
    start = clock();
    copyResultFromGPU();
    endt = clock();
    mem_transfer += endt - start;
    printf("Memory Transfer time in the GPU: %f seconds\n", mem_transfer / (double) CLOCKS_PER_SEC);
    printf("Computation time in the GPU: %f seconds\n", compute / (double) CLOCKS_PER_SEC);
    printf("Total time in the GPU: %f seconds\n", (mem_transfer + compute) / (double) CLOCKS_PER_SEC);

    /*
    for (int i=0; i < C_MD.dimension1; i++) {
       printf("\n");
        for (int j=0; j<C_MD.dimension2; j++)
            printf("%.2f ", C_CPU[i*C_MD.dimension2+j]);
    }
    */

    //printf("Comparing answers...\n");
    //compareHostAndGpuOutput();

    return 0;
}

// allocate and initialize A and B using a random number generator
void allocateAndInitializeAB() {

    size_t sizeofA = A_MD.dimension1 * A_MD.dimension2 * sizeof(float);
    A = (float*) malloc(sizeofA);

    srand(5);
    //srand(time(NULL));
    for (int i = 0; i < A_MD.dimension1; i++) {
        for (int j = 0; j < A_MD.dimension2; j++) {
            int index = i * A_MD.dimension2 + j;
            A[index] = (rand() % 1000) * 0.001;
        }
    }

    size_t sizeofB = B_MD.dimension1 * B_MD.dimension2 * sizeof(float);
    B = (float*) malloc(sizeofB);
    for (int i = 0; i < B_MD.dimension1; i++) {
        for (int j = 0; j < B_MD.dimension2; j++) {
            int index = i * B_MD.dimension2 + j;
            B[index] = (rand() % 1000) * 0.001;
        }
    }
}

// allocate memory in the GPU for all matrices, and copy A and B content from the host CPU memory to the GPU memory
void copyMatricesToGPU() {

    size_t sizeofA = A_MD.dimension1 * A_MD.dimension2 * sizeof(float);
    check_error(hipMalloc((void **) &A_GPU, sizeofA));
    check_error(hipMemcpy(A_GPU, A, sizeofA, hipMemcpyHostToDevice));

    size_t sizeofB = B_MD.dimension1 * B_MD.dimension2 * sizeof(float);
    check_error(hipMalloc((void **) &B_GPU, sizeofB));
    check_error(hipMemcpy(B_GPU, B, sizeofB, hipMemcpyHostToDevice));

    size_t sizeofC = C_MD.dimension1 * C_MD.dimension2 * sizeof(float);
    check_error(hipMalloc((void **) &C_GPU, sizeofC));
}

// copy results from C_GPU which is in GPU card memory to C_CPU which is in the host CPU for result comparison
void copyResultFromGPU() {
    size_t sizeofC = C_MD.dimension1 * C_MD.dimension2 * sizeof(float);
    C_CPU = (float*) malloc(sizeofC);
    check_error(hipMemcpy(C_CPU, C_GPU, sizeofC, hipMemcpyDeviceToHost));
}

// do a straightforward matrix-matrix multiplication in the CPU
// notice that this implementation can be massively improved in the CPU by doing proper cache blocking but we are
// not providing you the efficient CPU implementation as that reveals too much about the ideal GPU implementation
void computeCpuMMM() {

    // allocate the result matrix for the CPU computation
    size_t sizeofC = C_MD.dimension1 * C_MD.dimension2 * sizeof(float);
    C = (float*) malloc(sizeofC);

    // compute C[i][j] as the sum of A[i][k] * B[k][j] for all columns k of A
    for (int i = 0; i < A_MD.dimension1; i++) {
        int a_i = i * A_MD.dimension2;
        int c_i = i * C_MD.dimension2;
        for (int j = 0; j < B_MD.dimension2; j++) {
            int c_index = c_i + j;
            C[c_index] = 0;
            for (int k = 0; k < B_MD.dimension1; k++) {
                int a_index = a_i + k;
                int b_index = k * B_MD.dimension2 + j;
                C[c_index] += A[a_index] * B[b_index];
            }
        }
    }
}

__global__ void computeGPUMMM(float* A, float* B, float* C, int size) {

    // get the row and column of current thread
    // this row and column is relative to the top left
    // tile in this block's region
    int row = blockIdx.y * 3*BLOCK_DIM + threadIdx.y;
    int col = blockIdx.x * 3*BLOCK_DIM + threadIdx.x;

    // allocate this blocks quadrant (9X the block area)
    __shared__ float AA[3*BLOCK_DIM][3*BLOCK_DIM];
    __shared__ float BB[3*BLOCK_DIM][3*BLOCK_DIM];

    // sums for each point this thread responsible for
    float sum0 = 0;
    float sum1 = 0;
    float sum2 = 0;
    float sum3 = 0;
    float sum4 = 0;
    float sum5 = 0;
    float sum6 = 0;
    float sum7 = 0;
    float sum8 = 0;

    int kk;
    int k;
    int i;
    int j;
    
    // ^  15 local variables should fit into the 32 available registers

    // as all matrix qudrants necessary for this block's computation will not 
    // fit into shared memory, the entire computation is blocked by kk
    for (kk = 0; kk < size; kk += 3*BLOCK_DIM) {

        // load the approprate data from memory
        for (i = 0; i < 3; i++) {
            for (j = 0; j < 3; j++) {
                AA[threadIdx.y + i*BLOCK_DIM][threadIdx.x + j*BLOCK_DIM] =
                    A[(i*BLOCK_DIM + row) * size + threadIdx.x + kk + j*BLOCK_DIM];
                BB[threadIdx.y + i*BLOCK_DIM][threadIdx.x + j*BLOCK_DIM] =
                    B[(kk + threadIdx.y + i*BLOCK_DIM) * size + col + j*BLOCK_DIM];
            }
        }

        // since we stride across memory written by different warps, a sync is needed
        __syncthreads();

        // compute the partial dot products for all four quadrants
        for (k = 0; k < 3*BLOCK_DIM; k++) {
            sum0 += AA[threadIdx.y][k] * BB[k][threadIdx.x]; 
            sum1 += AA[threadIdx.y][k] * BB[k][threadIdx.x + BLOCK_DIM]; 
            sum2 += AA[threadIdx.y][k] * BB[k][threadIdx.x + 2*BLOCK_DIM]; 
            sum3 += AA[threadIdx.y + BLOCK_DIM][k] * BB[k][threadIdx.x]; 
            sum4 += AA[threadIdx.y + BLOCK_DIM][k] * BB[k][threadIdx.x + BLOCK_DIM]; 
            sum5 += AA[threadIdx.y + BLOCK_DIM][k] * BB[k][threadIdx.x + 2*BLOCK_DIM]; 
            sum6 += AA[threadIdx.y + 2*BLOCK_DIM][k] * BB[k][threadIdx.x]; 
            sum7 += AA[threadIdx.y + 2*BLOCK_DIM][k] * BB[k][threadIdx.x + BLOCK_DIM]; 
            sum8 += AA[threadIdx.y + 2*BLOCK_DIM][k] * BB[k][threadIdx.x + 2*BLOCK_DIM]; 
        }

        // sync is needed before writing to AA & BB
        __syncthreads();
    }

    // write the final dot products to C
    C[row * size + col] = sum0;
    C[row * size + col + BLOCK_DIM] = sum1;
    C[row * size + col + 2*BLOCK_DIM] = sum2;
    C[(row + BLOCK_DIM) * size + col] = sum3;
    C[(row + BLOCK_DIM) * size + col + BLOCK_DIM] = sum4;
    C[(row + BLOCK_DIM) * size + col + 2*BLOCK_DIM] = sum5;
    C[(row + 2*BLOCK_DIM) * size + col] = sum6;
    C[(row + 2*BLOCK_DIM) * size + col + BLOCK_DIM] = sum7;
    C[(row + 2*BLOCK_DIM) * size + col + 2*BLOCK_DIM] = sum8;

}

// function to determine if the GPU computation is done correctly by comparing the output from the GPU with that
// from the CPU
void compareHostAndGpuOutput() {
    int totalElements = C_MD.dimension1 * C_MD.dimension2;
    int missmatchCount = 0;
    for (int i = 0; i < totalElements; i++) {
        if (fabs(C[i] - C_CPU[i]) > 0.01) {
            missmatchCount++;
            //printf("mismatch at index %i: %f\t%f\n", i, C[i], C_CPU[i]);
        }
    }
    if (missmatchCount > 0) {
        printf("Computation is incorrect: outputs do not match in %d indexes\n", missmatchCount);
    } else {
        printf("Computation is correct: CPU and GPU outputs match\n");
    }
}

// Prints the specified error message and then exits
void die(const char *error) {
    printf("%s", error);
    exit(1);
}

// If the specified error code refers to a real error, report it and quit the program
void check_error(hipError_t e) {
    if (e != hipSuccess) {
        printf("\nCUDA error: %s\n", hipGetErrorString(e));
        exit(1);
    }
}

/*
void optimizedCpuMMM() {
    // allocate the result matrix for the CPU computation
    size_t sizeofC = C_MD.dimension1 * C_MD.dimension2 * sizeof(float);
    C = (float*) malloc(sizeofC);
    memset(C, 0f, sizeofC);

    // compute C[i][j] as the sum of A[i][k] * B[k][j] for all columns k of A
    int BLOCK_SIZE = 50;
    for (int jj = 0; jj < B_MD.dimension2; jj += BLOCK_SIZE) {
        int j_limit = jj + BLOCK_SIZE;
        for (int j = 0; j < B_MD.dimension1; j++) {
            int c_index = j;
            for (int ii = 0; i < A_MD.dimension1; ii += BLOCK_SIZE) {
                int i_limit = ii + BLOCK_SIZE;
                for (int i = 0; i < A_MD.dimension2; i++) {
                    int a_i = i * A_MD.dimension2;
                    int c_i += i * C_MD.dimension2;
                    for (int k = 0; k < BLOCK_SIZE; k++) {
                        int a_index = a_i + k + ;
                        int b_index = k * B_MD.dimension2 + j;
                        C[c_index] += A[a_index] * B[b_index];
                    }
                }
            }
        }
    }
}
*/
